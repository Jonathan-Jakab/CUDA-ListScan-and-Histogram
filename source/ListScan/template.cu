#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <wb.h>

#define BLOCK_SIZE 512 

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__global__ void scan(float *input, float *output, float *aux, int len) {
    //@@ Modify the body of this kernel to generate the scanned blocks
    //@@ Make sure to use the workefficient version of the parallel scan
    //@@ Also make sure to store the block sum to the aux array 

    __shared__ float XY[2 * BLOCK_SIZE];

    int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;

    if (i < len) {
        XY[threadIdx.x] = input[i];
    } else {
        XY[threadIdx.x] = 0.0;
    }
    if (i + blockDim.x < len) {
        XY[threadIdx.x + blockDim.x] = input[i + blockDim.x];
    } else {
        XY[threadIdx.x + blockDim.x] = 0.0;
    }

    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        __syncthreads();
        int index = (threadIdx.x + 1) * stride * 2- 1;
        if (index < 2 * BLOCK_SIZE) {
            XY[index] += XY[index - stride];
        }
    }

    for (int stride = BLOCK_SIZE / 2; stride > 0; stride /= 2) {
        __syncthreads();
        int index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index + stride < 2 * BLOCK_SIZE) {
            XY[index + stride] += XY[index];
        }
    }

    __syncthreads();

    if (i < len) {
        output[i] = XY[threadIdx.x];
    }
    if (i + blockDim.x < len) {
        output[i + blockDim.x] = XY[threadIdx.x + blockDim.x];
    }

    __syncthreads();

    if (threadIdx.x == BLOCK_SIZE - 1) {
        aux[blockIdx.x] = XY[2 * BLOCK_SIZE - 1];
    }
}

__global__ void addScannedBlockSums(float *output, float *aux, int len) {
	//@@ Modify the body of this kernel to add scanned block sums to 
	//@@ all values of the scanned blocks

    int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;

    if (blockIdx.x != 0 && i < len) {
        output[i] += aux[blockIdx.x - 1];
    }
    if (blockIdx.x != 0 && i + blockDim.x < len) {
        output[i + blockDim.x] += aux[blockIdx.x - 1];
    }
}

int main(int argc, char** argv) {
    wbArg_t args;
    float* hostInput;  // The input 1D list
    float* hostOutput; // The output 1D list
    float* deviceInput;
    float* deviceOutput;
    float* deviceAuxArray, * deviceAuxScannedArray;
    int numElements; // number of elements in the input/output list

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (float*)wbImport(wbArg_getInputFile(args, 0), &numElements);
    hostOutput = (float*)malloc(numElements * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numElements);

  wbTime_start(GPU, "Allocating device memory.");
  //@@ Allocate device memory
  //you can assume that aux array size would not need to be more than BLOCK_SIZE*2 (i.e., 1024)
  hipMalloc((void**)&deviceInput, numElements * sizeof(float));
  hipMalloc((void**)&deviceOutput, numElements * sizeof(float));
  hipMalloc((void**)&deviceAuxArray, ceil(numElements / (2.0 * BLOCK_SIZE)) * sizeof(float));
  hipMalloc((void**)&deviceAuxScannedArray, ceil(numElements / (2.0 * BLOCK_SIZE)) * sizeof(float));

  wbTime_stop(GPU, "Allocating device memory.");

  wbTime_start(GPU, "Clearing output device memory.");
  //@@ zero out the deviceOutput using hipMemset() by uncommenting the below line
  wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));

  wbTime_stop(GPU, "Clearing output device memory.");

  wbTime_start(GPU, "Copying input host memory to device.");
  //@@ Copy input host memory to device	
  hipMemcpy(deviceInput, hostInput, numElements * sizeof(float), hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input host memory to device.");

  //@@ Initialize the grid and block dimensions here
  dim3 dimGrid1(ceil(numElements / (2.0 * BLOCK_SIZE)), 1, 1);
  dim3 dimGrid2(1, 1, 1);
  dim3 dimBlock(BLOCK_SIZE, 1, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce
  //@@ You need to launch scan kernel twice: 1) for generating scanned blocks 
  //@@ (hint: pass deviceAuxArray to the aux parameter)
  //@@ and 2) for generating scanned aux array that has the scanned block sums. 
  //@@ (hint: pass NULL to the aux parameter)
  //@@ Then you should call addScannedBlockSums kernel.
  scan<<<dimGrid1, dimBlock>>>(deviceInput, deviceOutput, deviceAuxArray, numElements);
  scan<<<dimGrid2, dimBlock>>>(deviceAuxArray, deviceAuxScannedArray, deviceInput, ceil(numElements / (2.0 * BLOCK_SIZE)) );    //Note: tried setting the aux parameter to NULL but whenever I did that all solutions came out to -4.31602e+08 on everything. Using deviceInput seems to work fine.
  addScannedBlockSums<<<dimGrid1, dimBlock>>>(deviceOutput, deviceAuxScannedArray, numElements);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output device memory to host");
  //@@ Copy results from device to host	
  hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float), hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output device memory to host");

  wbTime_start(GPU, "Freeing device memory");
  //@@ Deallocate device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);
  hipFree(deviceAuxArray);
  hipFree(deviceAuxScannedArray);

  wbTime_stop(GPU, "Freeing device memory");

  wbSolution(args, hostOutput, numElements);

  free(hostInput);
  free(hostOutput);

  return 0;
}
